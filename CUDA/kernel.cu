#include "hip/hip_runtime.h"
#include <iostream>

#include "kernel.h"
#include "hip/hip_runtime.h"

void alloc_gpu(float *dst, int size) {
  CHECK(hipMalloc((float**)&dst, sizeof(float) * size));
}

void cpy_cuda(float *dst, float *src, int size, hipMemcpyKind kind) {
  CHECK(hipMemcpy(dst, src, sizeof(float) * size, kind));
}

void cpy_cpu2gpu(float *src, float *dst, int size) {
  CHECK(hipMemcpy(dst, src, sizeof(float) * size, hipMemcpyHostToDevice));
}

void cpy_gpu2cpu(float *src, float *dst, int size) {
  CHECK(hipMemcpy((void**)&dst, (void**)&src, sizeof(float) * size, hipMemcpyDeviceToHost));
}

__global__ void sum_gpu(float *a, float *b, float *c, int size) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= size) {
    return ;
  }
  c[idx] = a[idx] + b[idx];
}

void sum(float *a, float *b, float *c, int size) {
  dim3 block(32);
  dim3 grid((size + block.x - 1) / block.x);
  sum_gpu <<< grid, block >>> (a, b, c, size);
  CHECK(hipDeviceSynchronize());
}
